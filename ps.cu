// vAdd.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256
 
__global__ void getDensity (double *a_d, double *b_d, int numSlice)
{
     // __shared__ double s[BLOCK_SIZE + 2];
      int x = blockIdx.x * blockDim.x + threadIdx.x;

      

      if (x != 0 && x < numSlice - 1) {
        // if(threadIdx.x == 0) {
        // s[0] = a_d[x-1];
        // s[1] = a_d[x];
        // }
        // else if (threadIdx.x == BLOCK_SIZE - 1){
        //   s[BLOCK_SIZE +1] = a_d[x + 1];
        //   s[BLOCK_SIZE] = a_d[x];
        // }
        // else {
        //   s[threadIdx.x + 1] = a_d[x];
        // }
        b_d[x] = (a_d[x-1] + a_d[x+1])/2;
      }
      __syncthreads();
      if (x == numSlice - 1){
      	b_d[numSlice - 1] = b_d[numSlice - 2];
      }

    // Copying result with the neighboring values, just to check the neighbor value

}

__global__ void initialize (double *a_d, double *b_d,  int numSlice, int pointSource)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (x == 0){
		a_d[x] = pointSource;
		b_d[x] = pointSource;
	}
	else if (x < numSlice) {
		a_d[x] = 0;
		b_d[x] = 0;
	}
}


extern "C" double compute(int numSlice, int time, double densityAt, int pointSource)
{
	double *a_d, *b_d;

	hipMalloc ((void**) &a_d, sizeof(double) * numSlice);
	hipMalloc ((void**) &b_d, sizeof(double) * numSlice);
	//cudaMalloc ((void**) &c_d, sizeof(double) * 5);
	double density = 0;
	
	initialize <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, numSlice, pointSource);

  for (int n = 0; n < time; n++){
    getDensity <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, numSlice);
    a_d = b_d;
  }

  int index = (int)((numSlice - 1) * 0.7);


  hipMemcpy(&density, &a_d[index], sizeof(double), hipMemcpyDeviceToHost);


	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipFree (a_d);
	hipFree (b_d);

  return density;
}

