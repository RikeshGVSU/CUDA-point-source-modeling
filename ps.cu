// vAdd.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256
 
__global__ void getDensity (double *a_d, double *b_d, double *c_d, int numSlice, int time, double densityAt, int pointSource, double density_d)
{
   	//double * temp;
  int x = blockIdx.x * blockDim.x + threadIdx.x;	
  for (int n = 0; n < 1000; n++){
      if (x != 0 && x < numSlice - 1) {
        b_d[x] = (a_d[x - 1] + a_d[x + 1])/2;
      }
      __syncthreads();
      if (x == numSlice - 1){
      	b_d[x] = b_d[x - 1];
      }
      n++;
      __syncthreads();
      if (x != 0 && x < numSlice - 1) {
        a_d[x] = (b_d[x - 1] + b_d[x + 1])/2;
      }
      __syncthreads();
      if (x == numSlice - 1){
      	a_d[x] = a_d[x - 1];
      }
    }

    // Copying result with the neighboring values, just to check the neighbor value

    int index = (int)((numSlice - 1) * 0.7);
    c_d[0] =  a_d[index - 2];
    c_d[1] =  a_d[index - 1];
    c_d[2] =  a_d[index];
    c_d[3] =  a_d[index+2];
    c_d[4] =  a_d[index+1];


}

__global__ void initialize (double *a_d, double *b_d,  int numSlice, int pointSource)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (x == 0){
		a_d[x] = pointSource;
		b_d[x] = pointSource;
	}
	else if (x < numSlice) {
		a_d[x] = 0;
		b_d[x] = 0;
	}
}

extern "C" void compute(double *c, int numSlice, int time, double densityAt, int pointSource)
{
	double *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(double) * numSlice);
	hipMalloc ((void**) &b_d, sizeof(double) * numSlice);
	hipMalloc ((void**) &c_d, sizeof(double) * 5);
	double density_d = 0;
	
	initialize <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, numSlice, pointSource);

	getDensity <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, c_d, numSlice, time, densityAt, pointSource, density_d);

	hipMemcpy (c, c_d, sizeof(double) * 5, hipMemcpyDeviceToHost);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
}

