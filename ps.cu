// vAdd.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256
 
__global__ void getDensity (double *a_d, double *b_d, int numSlice)
{
   	//double * temp;
      int x = blockIdx.x * blockDim.x + threadIdx.x;	
      if (x != 0 && x < numSlice - 1) {
        b_d[x] = (a_d[x - 1] + a_d[x + 1])/2;
      }
      __syncthreads();
      if (x == numSlice - 1){
      	b_d[numSlice - 1] = b_d[numSlice - 2];
      }

    // Copying result with the neighboring values, just to check the neighbor value

}

__global__ void initialize (double *a_d, double *b_d,  int numSlice, int pointSource)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (x == 0){
		a_d[x] = pointSource;
		b_d[x] = pointSource;
	}
	else if (x < numSlice) {
		a_d[x] = 0;
		b_d[x] = 0;
	}
}

extern "C" double compute(int numSlice, int time, double densityAt, int pointSource)
{
	double *a_d, *b_d, *temp;

	hipMalloc ((void**) &a_d, sizeof(double) * numSlice);
	hipMalloc ((void**) &b_d, sizeof(double) * numSlice);
	//cudaMalloc ((void**) &c_d, sizeof(double) * 5);
	double density = 0;
	
	initialize <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, numSlice, pointSource);

  for (int n = 0; n < time; n++){
    getDensity <<< ceil((float) numSlice/BLOCK_SIZE), BLOCK_SIZE>>> (a_d, b_d, numSlice);
    temp = a_d;
    a_d = b_d;
    b_d = temp;
  }

  int index = (int)((numSlice - 1) * 0.7);


  hipMemcpy(&density, &a_d[index], sizeof(double), hipMemcpyDeviceToHost);


	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipFree (a_d);
	hipFree (b_d);

  return density;
}

